#include "hip/hip_runtime.h"
/*============================================================================
 * Dispatch test, CUDA implementations.
 *============================================================================*/

/*
  This file is part of code_saturne, a general-purpose CFD tool.

  Copyright (C) 1998-2024 EDF S.A.

  This program is free software; you can redistribute it and/or modify it under
  the terms of the GNU General Public License as published by the Free Software
  Foundation; either version 2 of the License, or (at your option) any later
  version.

  This program is distributed in the hope that it will be useful, but WITHOUT
  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
  FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
  details.

  You should have received a copy of the GNU General Public License along with
  this program; if not, write to the Free Software Foundation, Inc., 51 Franklin
  Street, Fifth Floor, Boston, MA 02110-1301, USA.
*/

/*----------------------------------------------------------------------------*/

#include "cs_defs.h"

#include <iostream>

/*----------------------------------------------------------------------------
 *  Header for the current file
 *----------------------------------------------------------------------------*/

#include "cs_dispatch.h"

/*! \cond DOXYGEN_SHOULD_SKIP_THIS */

/*----------------------------------------------------------------------------*/

/*=============================================================================
 * Local macros
 *============================================================================*/

/*=============================================================================
 * Local definitions
 *============================================================================*/

/*============================================================================
 * Private function definitions
 *============================================================================*/

/*! (DOXYGEN_SHOULD_SKIP_THIS) \endcond */

/*=============================================================================
 * Semi-private function definitions
 *============================================================================*/

/*----------------------------------------------------------------------------
 * Test dispatch class.
 *----------------------------------------------------------------------------*/

void
cs_dispatch_test_cuda(void)
{
  const cs_lnum_t n = 100;

  int device_id;
  hipGetDevice(&device_id);

  hipStream_t stream;
  hipStreamCreate(&stream);

  csContext ctx(csCudaContext(1, 1, stream, device_id), {});

  csCudaContext ctx_cuda = static_cast<csCudaContext&>(ctx);

  cs_real_t *a0, *a1;
  cs_alloc_mode_t amode = CS_ALLOC_HOST_DEVICE_SHARED;
  CS_MALLOC_HD(a0, n, cs_real_t, amode);
  CS_MALLOC_HD(a1, n, cs_real_t, amode);

  unsigned int blocksize = 64;
  unsigned int gridsize
    = (unsigned int)ceil((double)n / blocksize);

  ctx_cuda.set_cuda_config(gridsize, blocksize);
  ctx_cuda.iter(n, CS_HOST_DEVICE_FUNCTOR(=, (cs_lnum_t ii), {
    cs_lnum_t c_id = ii;
#ifdef __CUDA_ARCH__   // Test to know whether we are on GPU or CPU...
    a0[ii] = c_id*0.1;
#else
    a0[ii] = c_id;
#endif
    a1[ii] = cos(a0[ii]);
  }));

  hipStreamSynchronize(stream);

  for (cs_lnum_t ii = 0; ii < n; ii++) {
    std::cout << ii << " " << a0[ii] << " " << a1[ii] << std::endl;
  }

  std::cout << "device_id " << device_id << " " << gridsize << " " << blocksize << std::endl;

  CS_FREE_HD(a0);
  CS_FREE_HD(a1);
}
